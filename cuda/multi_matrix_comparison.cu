#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "TimeWatch.hpp"

#define DEBUG 1
#define MAT_SIZE 2000
#define BLOCK_SIZE 16
#define NUM_THREADS 8

int A[MAT_SIZE][MAT_SIZE];
int B[MAT_SIZE][MAT_SIZE];
int C[MAT_SIZE][MAT_SIZE];
int matrix_size;

TimeWatch time_watch;

void *thread(void *vargp);
__global__ void MatMulKernel(const int*, const int*, int*);
__global__ void MatMulKernelShare(int*, int*, int*);


void initMatrix();
void dotMultiplyMatrix();
void multiplyMatrix();
void printMatrix();

void printTime(char* s){
	printf("%s: %.4lf\n", s, time_watch.getElapsedTime());
}

int main(void){

	// printf("Input matrix size: ");
	// scanf("%d\n", &matrix_size);
	initMatrix();

	/*   Using single thread CPU
	*/

	time_watch.startCount();
	multiplyMatrix();
	printTime("Multiply using CPU");
	// printMatrix();

	/*	Using multiple threads CPU
	*/
	initMatrix();
	time_watch.startCount();	
	pthread_t tid[NUM_THREADS];
	for (int i = 0; i < NUM_THREADS; i++){
		int startLine = MAT_SIZE / NUM_THREADS * i;
		pthread_create(&tid[i], NULL, thread, (void *)startLine);
	}
	
	for (int i = 0; i < NUM_THREADS; i++)
		pthread_join(tid[i], NULL);

	printTime("Multiply using Multiple CPU");
	// printMatrix();	

	/* Using GPU
	*/

	size_t size = MAT_SIZE * MAT_SIZE * sizeof(int);
	int* d_A, *d_B;
	int* d_C;

	/*	Using GPU without Shared Memory
	*/

	time_watch.startCount();

	// Allocate A
	hipError_t err = hipMalloc((void**)&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_A, *A, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// Allocate B
	err = hipMalloc((void**)&d_B, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));	

	// Allocate C
	err = hipMalloc((void**)&d_C, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));

	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((MAT_SIZE + dimBlock.x - 1) / dimBlock.x, 
		(MAT_SIZE + dimBlock.y - 1) / dimBlock.y);
	
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Copy to host
	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printf("Copy C from device: %s\n", hipGetErrorString(err));
	
	printTime("Multiply Using GPU without Shared Memory");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// printMatrix();

	/*	Using GPU with Shared Memory
	*/

	time_watch.startCount();

	err = hipMalloc((void**)&d_A, size);
	printf("CUDA malloc A: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_A, *A, size, hipMemcpyHostToDevice);
	printf("Copy A to device: %s\n", hipGetErrorString(err));

	// Allocate B
	err = hipMalloc((void**)&d_B, size);
	printf("CUDA malloc B: %s\n", hipGetErrorString(err));
	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	printf("Copy B to device: %s\n", hipGetErrorString(err));	

	// Allocate C
	err = hipMalloc((void**)&d_C, size);
	printf("CUDA malloc C: %s\n", hipGetErrorString(err));

	// Invoke kernel
	dim3 dimBlock_Shared(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid_Shared(MAT_SIZE / dimBlock.x, MAT_SIZE / dimBlock.y);
	
	MatMulKernelShare<<<dimGrid_Shared, dimBlock_Shared>>>(d_A, d_B, d_C);

	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Copy to host
	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printf("Copy C from device: %s\n", hipGetErrorString(err));
	
	printTime("Multiply Using GPU with Shared Memory");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// printMatrix();

	return 0;

}

void initMatrix(){
	for (int i = 0; i < MAT_SIZE; i++)
		for (int j = 0; j < MAT_SIZE; j++){
			A[i][j] = i;
			B[i][j] = j;
			C[i][j] = 0;
		}
}

void dotMultiplyMatrix(){
	for (int i = 0; i < MAT_SIZE; i++)
		for (int j = 0; j < MAT_SIZE; j++){
			C[i][j] = A[i][j] * B[i][j]; 
		}
}

void multiplyMatrix(){
	for (int i = 0; i < MAT_SIZE; i++)
		for (int j = 0; j < MAT_SIZE; j++)
			for (int k = 0; k < MAT_SIZE; k++)
				C[i][j] += A[i][k] * B[k][j];
}

void printMatrix(){
	for (int i = 0; i < MAT_SIZE; i++){
		for (int j = 0; j < MAT_SIZE; j++){
			printf("%6d ", C[i][j]);
		}
		printf("\n");
	}
}

void *thread(void *vargp){
	int startLine = ((int)vargp);
	for (int i = startLine; i < startLine + MAT_SIZE / NUM_THREADS; i++)
		for (int j = 0; j < MAT_SIZE; j++)
			for (int k = 0; k < MAT_SIZE; k++)
				C[i][j] += A[i][k] * B[k][j];

	return NULL;
}


__global__ void MatMulKernel(const int* A, const int* B, int* C){
	// Each thread computes one element of C
	// by accumulatig results into Cvalue
	int Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= MAT_SIZE || col >= MAT_SIZE)
		return;
	for (int i = 0; i < MAT_SIZE; ++i)
		Cvalue += A[row * MAT_SIZE + i] * B[i * MAT_SIZE + col];

	C[row * MAT_SIZE + col] = Cvalue;
}

__device__ int GetElement(const int* A, int row, int col){
	return A[row * MAT_SIZE + col];
}

__device__ void SetElement(int* A, int row, int col, int value){
	A[row * MAT_SIZE + col] = value;
}

__device__ int* GetSubMatrix(int* A, int row, int col){
	int* Asub;
	Asub = &A[MAT_SIZE * BLOCK_SIZE * row + BLOCK_SIZE * col];

	return Asub;
}

__global__ void MatMulKernelShare(int* A, int* B, int* C){
	int blockRow = blockIdx.y, blockCol = blockIdx.x;
	int* Csub = GetSubMatrix(C, blockRow, blockCol);

	float Cvalue = 0.0;
	int row = threadIdx.y, col = threadIdx.x;

	for (int m = 0; m < (MAT_SIZE / BLOCK_SIZE); ++m){
		int* Asub = GetSubMatrix(A, blockRow, m);
		int* Bsub = GetSubMatrix(B, blockCol, m);

		__shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);

		__syncthreads();

		for (int e = 0; e < BLOCK_SIZE; ++e)
			Cvalue += As[row][e] * Bs[e][col];

		__syncthreads();
	}

	SetElement(Csub, row, col, Cvalue);

}